#include "hip/hip_runtime.h"
#include "cfunc_fourierrec.cuh"
#include "kernels_fourierrec.cuh"

cfunc_fourierrec::cfunc_fourierrec(size_t nproj, size_t nz, size_t n)
    : nproj(nproj), nz(nz), n(n) {
    float eps = 1e-3;
    mu = -log(eps) / (2 * n * n);        
    m = ceil(2 * n * 1 / PI * sqrt(-mu * log(eps) + (mu * n) * (mu * n) / 4));    
    hipMalloc((void **)&fde,
            (2 * n + 2 * m) * (2 * n + 2 * m) * nz * sizeof(real2));
    hipMalloc((void **)&x, n * nproj * sizeof(float));
    hipMalloc((void **)&y, n * nproj * sizeof(float));
    
    long long ffts[] = {2*n,2*n};
	  long long idist = (2 * n + 2 * m) * (2 * n + 2 * m);long long odist = (2 * n + 2 * m) * (2 * n + 2 * m);
    long long inembed[] = {2 * n + 2 * m, 2 * n + 2 * m};long long onembed[] = {2 * n + 2 * m, 2 * n + 2 * m};
    size_t workSize = 0;

    hipfftCreate(&plan2d);
    hipfftXtMakePlanMany(plan2d, 
        2, ffts, 
        inembed, 1, idist, CUDA_C, 
        onembed, 1, odist, CUDA_C, 
        nz, &workSize, CUDA_C);    
    // fft 1d
    hipfftCreate(&plan1d);
    ffts[0] = n;
    idist = n;
    odist = n;
    inembed[0] = n;
    onembed[0] = n;
    hipfftXtMakePlanMany(plan1d, 
        1, ffts, 
        inembed, 1, idist, CUDA_C, 
        onembed, 1, odist, CUDA_C, 
        nproj*nz, &workSize, CUDA_C);                      
        
  }


// destructor, memory deallocation
cfunc_fourierrec::~cfunc_fourierrec() { free(); }

void cfunc_fourierrec::free() {
  if (!is_free) {
    hipFree(fde);
    hipFree(x);
    hipFree(y);
    hipfftDestroy(plan2d);
    hipfftDestroy(plan1d);
    is_free = true;   
  }
}

void cfunc_fourierrec::backprojection(size_t f_, size_t g_, size_t theta_, size_t stream_) {
    real2* g = (real2 *)g_;    
    real2* f = (real2 *)f_;
    theta = (float*)theta_;
    hipStream_t stream = (hipStream_t)stream_;    
    hipfftSetStream(plan1d, stream);
    hipfftSetStream(plan2d, stream);    

    // set thread block, grid sizes will be computed before cuda kernel execution
    dim3 dimBlock(32,32,1);    
    dim3 GS2d0,GS3d0,GS3d1,GS3d2,GS3d3;  
    GS2d0 = dim3(ceil(n / 32.0), ceil(nproj / 32.0));
    GS3d0 = dim3(ceil(n / 32.0), ceil(n / 32.0),nz);
    GS3d1 = dim3(ceil(2 * n / 32.0), ceil(2 * n / 32.0),nz);
    GS3d2 = dim3(ceil((2 * n + 2 * m) / 32.0),ceil((2 * n + 2 * m) / 32.0), nz);
    GS3d3 = dim3(ceil(n / 32.0), ceil(nproj / 32.0),nz);
   
    
    hipMemsetAsync(fde, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * nz * sizeof(real2),stream);
    
    takexy <<<GS2d0, dimBlock, 0, stream>>> (x, y, theta, n, nproj);        
    ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, nproj, nz);
    hipfftXtExec(plan1d, g, g, HIPFFT_FORWARD);
    ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, nproj, nz);    
    mulc <<<GS3d3, dimBlock, 0, stream>>> (g, 4/(float)n, n, nproj, nz);
    
    gather <<<GS3d3, dimBlock, 0, stream>>> (g, fde, x, y, m, mu, n, nproj, nz, 0);    
    
    wrap <<<GS3d2, dimBlock, 0, stream>>> (fde, n, nz, m, 0);
    
    fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fde, 2 * n + 2 * m, nz);
    hipfftXtExec(plan2d, &fde[m + m * (2 * n + 2 * m)],
               &fde[m + m * (2 * n + 2 * m)], HIPFFT_BACKWARD);
    fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fde, 2 * n + 2 * m, nz);
    
    divphi <<<GS3d0, dimBlock, 0, stream>>> (fde, f, mu, n, nz, nproj, m, 0);        
    circ <<<GS3d0, dimBlock, 0, stream>>> (f, 0, n, nz);  
}


void cfunc_fourierrec::projection(size_t g_, size_t f_, size_t theta_, size_t stream_) {
  real2* g = (real2 *)g_;    
  real2* f = (real2 *)f_;
  theta = (float*)theta_;
  hipStream_t stream = (hipStream_t)stream_;    
  hipfftSetStream(plan1d, stream);
  hipfftSetStream(plan2d, stream);    

  // set thread block, grid sizes will be computed before cuda kernel execution
  dim3 dimBlock(32,32,1);    
  dim3 GS2d0,GS3d0,GS3d1,GS3d2,GS3d3;  
  GS2d0 = dim3(ceil(n / 32.0), ceil(nproj / 32.0));
  GS3d0 = dim3(ceil(n / 32.0), ceil(n / 32.0),nz);
  GS3d1 = dim3(ceil(2 * n / 32.0), ceil(2 * n / 32.0),nz);
  GS3d2 = dim3(ceil((2 * n + 2 * m) / 32.0),ceil((2 * n + 2 * m) / 32.0), nz);
  GS3d3 = dim3(ceil(n / 32.0), ceil(nproj / 32.0),nz);
 
  
  hipMemsetAsync(fde, 0, (2 * n + 2 * m) * (2 * n + 2 * m) * nz * sizeof(real2),stream);  
  
  takexy <<<GS2d0, dimBlock, 0, stream>>> (x, y, theta, n, nproj);        
  circ <<<GS3d0, dimBlock, 0, stream>>> (f, 0, n, nz);  
  divphi <<<GS3d0, dimBlock, 0, stream>>> (fde, f, mu, n, nz, nproj, m, 1);        
  fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fde, 2 * n + 2 * m, nz);
  hipfftXtExec(plan2d, &fde[m + m * (2 * n + 2 * m)],
             &fde[m + m * (2 * n + 2 * m)], HIPFFT_FORWARD);
  fftshiftc <<<GS3d2, dimBlock, 0, stream>>> (fde, 2 * n + 2 * m, nz);
  wrap <<<GS3d2, dimBlock, 0, stream>>> (fde, n, nz, m, 1);
  gather <<<GS3d3, dimBlock, 0, stream>>> (g, fde, x, y, m, mu, n, nproj, nz, 1);    
  mulc <<<GS3d3, dimBlock, 0, stream>>> (g, 4/(float)n, n, nproj, nz);

  ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, nproj, nz);
  hipfftXtExec(plan1d, g, g, HIPFFT_BACKWARD);
  ifftshiftc <<<GS3d3, dimBlock, 0, stream>>> (g, n, nproj, nz);      
}
