#include "hip/hip_runtime.h"
#include "cfunc_lprec.cuh"
#include "kernels_lprec.cuh"
#include<stdio.h>
hipError_t copy3DDeviceToArray(hipArray* dfa, real* df, hipExtent ext, hipStream_t stream)
{
	hipMemcpy3DParms param = { 0 };
	param.srcPtr   = make_hipPitchedPtr((void*)df, ext.width*sizeof(real), ext.width, ext.height);
	param.dstArray = dfa;
	param.kind = hipMemcpyDeviceToDevice;
	param.extent = ext;
	return hipMemcpy3DAsync(&param,stream);
}

cfunc_lprec::cfunc_lprec(int nproj, int nz, int n, int ntheta, int nrho):
nproj(nproj), nz(nz), n(n), ntheta(ntheta), nrho(nrho) {

    // Create FFT plans for Fourier Transform in log-polar coordinates
    long long ffts[] = {nrho,ntheta};
	long long idist = nrho*ntheta;long long odist = nrho*(ntheta/2+1);
    long long inembed[] = {nrho, ntheta};long long onembed[] = {nrho, ntheta/2+1};
    size_t workSize = 0;
    hipfftCreate(&plan_forward);
    hipfftXtMakePlanMany(plan_forward, 
        2, ffts, 
        inembed, 1, idist, CUDA_R, 
        onembed, 1, odist, CUDA_C, 
        nz, &workSize, CUDA_C);    
    hipfftCreate(&plan_inverse);
    hipfftXtMakePlanMany(plan_inverse, 
        2, ffts, 
        onembed, 1, odist, CUDA_C, 
        inembed, 1, idist, CUDA_R, 
        nz, &workSize, CUDA_R);
    
    // Allocate temporarily arrays 
    hipMalloc((void **)&fl, nz*ntheta*nrho*sizeof(real)); 
    hipMalloc((void **)&flc, nz*(ntheta/2+1)*nrho*sizeof(real2)); 
    hipMalloc((void **)&gtmp, nz*n*nproj*sizeof(real)); 
    
    // 3d arrays for textures
    hipChannelFormatDesc texf_desc = CUDA_CREATE_CHANNEL_DESC();    
    hipError_t b = hipMalloc3DArray(&ga, &texf_desc, make_hipExtent(n,nproj,nz),hipArrayLayered);    
    hipMalloc3DArray(&fla, &texf_desc, make_hipExtent(ntheta,nrho,nz),hipArrayLayered);
    
    // texture objects
    hipTextureDesc             texDescr;    
    memset(&texDescr,0,sizeof(hipTextureDesc));
    
	texDescr.addressMode[0] = hipAddressModeWrap;
	texDescr.addressMode[1] = hipAddressModeWrap;
	texDescr.filterMode = hipFilterModeLinear;
    texDescr.normalizedCoords = true;
    texDescr.readMode = hipReadModeElementType;
    
    hipResourceDesc texRes;    
    memset(&texRes,0,sizeof(hipResourceDesc));
    texRes.resType            = hipResourceTypeArray;    
    texRes.res.array.array    = ga;    
    hipCreateTextureObject(&texg, &texRes, &texDescr, NULL);    
    texRes.res.array.array    = fla;
    hipCreateTextureObject(&texfl, &texRes, &texDescr, NULL);    
    is_free = false;    
}

// destructor, memory deallocation
cfunc_lprec::~cfunc_lprec() { free(); }

void cfunc_lprec::free() {
    if (!is_free) {
        hipfftDestroy(plan_forward);
        hipfftDestroy(plan_inverse);    
        hipDestroyTextureObject(texg);
        hipDestroyTextureObject(texfl);
        hipFree(fl);
        hipFree(flc);
        hipFree(gtmp);
        hipFreeArray(ga);        
        hipFreeArray(fla);

        is_free = true;
    }
}

void cfunc_lprec::setgrids(size_t fz_, size_t lp2p1_, size_t lp2p2_, size_t lp2p1w_, size_t lp2p2w_, 
    size_t C2lp1_, size_t C2lp2_, size_t lpids_, size_t wids_, size_t cids_, 
    size_t nlpids_, size_t nwids_, size_t ncids_){
        
    fz = (real2*)fz_;
    lp2p1 = (float*)lp2p1_;
    lp2p2 = (float*)lp2p2_;
    lp2p1w = (float*)lp2p1w_;
    lp2p2w = (float*)lp2p2w_;
    C2lp1 = (float*)C2lp1_;
    C2lp2 = (float*)C2lp2_;
    lpids = (int*)lpids_;
    wids = (int*)wids_;
    cids = (int*)cids_;
    nlpids = nlpids_;
    nwids = nwids_;
    ncids = ncids_;        
}

void cfunc_lprec::backprojection(size_t f_, size_t g_, size_t stream_) 
{
    real* f = (real*)f_;
    real* g = (real*)g_;
    hipStream_t stream = (hipStream_t)stream_;    
    hipfftSetStream(plan_forward, stream);
    hipfftSetStream(plan_inverse, stream);    
    
    hipMemsetAsync(f, 0, nz*n*n*sizeof(real),stream); 

    // set thread block, grid sizes will be computed before cuda kernel execution
    dim3 dimBlock(BS1,BS2,BS3);    
    uint GS1, GS2, GS3;    
    
    // define all block and grid sizes
    GS1 = (uint)ceil(n/(float)BS1); GS2 = (uint)ceil(nproj/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid01(GS1,GS2,GS3);    
	GS1 = (uint)ceil(nproj/(float)BS1);GS2 = (uint)ceil(nz/(float)BS2); dim3 dimGrid02(GS1,GS2,1);    	
	GS1 = (uint)ceil(nproj/(float)BS1);GS2 = (uint)ceil(n/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3); dim3 dimGrid03(GS1,GS2,GS3);    	
	GS1 = (uint)ceil(n/(float)BS1);GS2 = (uint)ceil(nz/(float)BS2); dim3 dimGrid04(GS1,GS2,1); 
	GS1 = (uint)ceil(ceil(sqrt(nlpids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(nlpids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid1(GS1,GS2,GS3);int step2d1 = BS1*GS1;  
    GS1 = (uint)ceil(ceil(sqrt(nwids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(nwids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid2(GS1,GS2,GS3);int step2d2 = BS1*GS1;      
    GS1 = (uint)ceil((ntheta/2+1)/(float)BS1); GS2 = (uint)ceil(nrho/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid3(GS1,GS2,GS3);int step2d3 = BS1*GS1;      
    GS1 = (uint)ceil(ceil(sqrt(ncids))/(float)BS1); GS2 = (uint)ceil(ceil(sqrt(ncids))/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3);dim3 dimGrid4(GS1,GS2,GS3);int step2d4 = BS1*GS1;  	
    GS1 = (uint)ceil(n/(float)BS1);GS2 = (uint)ceil(n/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3); dim3 dimGrid11(GS1,GS2,GS3); 
    GS1 = (uint)ceil(ntheta/(float)BS1);GS2 = (uint)ceil(nrho/(float)BS2);GS3 = (uint)ceil(nz/(float)BS3); dim3 dimGrid12(GS1,GS2,GS3); 

    ////// Prefilter for cubic interpolation in polar coordinates //////
	//transpose for optimal cache usage
	transpose<<<dimGrid01,dimBlock, 0, stream>>>(gtmp, g,n, nproj,nz);
	//compensate in samples for x direction
	SamplesToCoefficients2DY<<<dimGrid02, dimBlock, 0, stream>>>(gtmp,nproj*sizeof(real),nproj, n,nz);
	// //transpose back
	transpose<<<dimGrid03,dimBlock, 0, stream>>>(g,gtmp,nproj, n,nz);
	//compensate in samples for y direction
	SamplesToCoefficients2DY<<<dimGrid04, dimBlock, 0, stream>>>(g,n*sizeof(real),n,nproj,nz);    
    //copy to the array associated with texture memory
    copy3DDeviceToArray(ga,g,make_hipExtent(n, nproj, nz),stream);
    
    //////// Iterations over log-polar angular spans ///////    	
    for(int k=0; k<3;k++)
    {
        hipMemsetAsync(fl, 0, nz*ntheta*nrho*sizeof(real),stream); 
		//interp from polar to log-polar grid
        interp<<<dimGrid1, dimBlock, 0, stream>>>(texg, fl,&lp2p2[k*nlpids],&lp2p1[k*nlpids],step2d1,nlpids,n,nproj,nz,lpids,ntheta*nrho);
		//interp from polar to log-polar grid additional points
        interp<<<dimGrid2, dimBlock, 0, stream>>>(texg, fl,&lp2p2w[k*nwids],&lp2p1w[k*nwids],step2d2,nwids,n,nproj,nz,wids,ntheta*nrho);
        //Forward FFT
        hipfftXtExec(plan_forward, fl,flc,HIPFFT_FORWARD);        
		//multiplication by adjoint transfer function and division by FFT of the cubic spline in log-polar coordinates (fz:=:fz/fB3)
        mul<<<dimGrid3, dimBlock, 0, stream>>>(flc,fz,ntheta/2+1,nrho,nz);
		//Inverse FFT
        hipfftXtExec(plan_inverse,flc,fl,HIPFFT_BACKWARD);        
        mulc<<<dimGrid12,dimBlock,0,stream>>>(fl, 2/(float)(nrho*ntheta), ntheta, nrho, nz);
        // //copy to binded texture 
        copy3DDeviceToArray(fla,fl,make_hipExtent(ntheta, nrho, nz),stream);
        // //interp from log-polar to Cartesian grid
        interp<<<dimGrid4, dimBlock, 0, stream>>>(texfl, f,&C2lp1[k*ncids],&C2lp2[k*ncids],step2d4,ncids,ntheta,nrho,nz,cids,n*n);                    
    }
    
}